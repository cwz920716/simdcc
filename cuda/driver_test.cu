#include <stdio.h>
#include <opencv2/opencv.hpp>

using namespace cv;

int main(int argc, char **argv) {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }

  if (argc != 2) {
      printf("usage: driver_test <Image_Path>\n");
      return -1;
  }

  Mat image;
  image = imread( argv[1], 1 );
  if (!image.data) {
    printf("No image data \n");
    return -1;
  }
  namedWindow("Display Image", WINDOW_AUTOSIZE );
  imshow("Display Image", image);
  waitKey(0);
  return 0;
}
