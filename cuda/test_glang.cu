#include "hip/hip_runtime.h"
#include "glog/logging.h"
#include "synthesis/base.h"
#include "synthesis/iteratable.h"
#include "synthesis/expand.h"
#include "synthesis/for.h"
#include "synthesis/gbar.cuh"
#include "synthesis/pool.h"
#include "cuda_util.h"

__global__ void type_test(glang::DynArray<float> out, glang::Slice S) {
  if (threadIdx.x == 0) {
    printf("[%d:%d:%d]\n", S.start(), S.end(), S.step());
    // printf("%p[%d:%d:%d]\n", out.data(), out.start(), out.end(), out.step());
  }

  auto f = [&] DEVICE (float &x) {
    x = threadIdx.x;
    // printf("x[%p]=%f\n", &x, x);
  };

  glang::Expand<float, glang::kBlockScope> expand;
  expand(out, f);

  glang::Parfor<float, glang::kBlockScope> parfor;
  parfor(out, f);
}

template<int NUM_THREADS>
__global__ void spmv_test(glang::DynArray<int> R, glang::DynArray<float> M) {
  typedef glang::BlockPool<glang::kBlockScope, NUM_THREADS> BlockPool;
  __shared__ typename BlockPool::TempStorage temp_storage;
  BlockPool pool(temp_storage);

  glang::Slice RSlice(R.start(), R.end());
  auto Rc = R;

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  auto visit = [&] DEVICE (int i) {
    printf("[%d] visit col %d\n", tid, i);
  };

  auto f = [&] DEVICE (int i) {
    glang::Slice S(Rc[i], Rc[i+1]);
    int r, t;
    pool.claim(Rc[i + 1] - Rc[i], r, t);
    printf("[%d] visit row %d[%d:%d:%d] alloc resources at [%d<-%d]\n",
        tid, i, S.start(), S.end(), S.step(), r, t);

    glang::ForEach<int> for_each;
    for_each(S, visit);
  };

  glang::Expand<int, glang::kDeviceScope> expand;
  expand(RSlice, f);
}

template<int NUM_THREADS>
__global__ void pool_test() {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.x, j, t;
  typedef glang::BlockPool<glang::kWarpScope, NUM_THREADS> BlockPool;
  __shared__ typename BlockPool::TempStorage temp_storage;
  BlockPool(temp_storage).claim(i, j, t);
  printf("[%d] alloc %d at %d\n", tid, i, j);
}

template<class T>
glang::DynArray<T> makeDA(int len, T *hData, T default_v) {
  T *dData;
  CUDA_CHECK(hipMalloc(&dData, len * sizeof(T)));
  CUDA_CHECK(hipMemcpy(dData, hData, len * sizeof(T), hipMemcpyHostToDevice));
  return glang::DynArray<T>(len, dData, default_v);
}

int main(void) {
  const int kDataLen = 64;
  glang::Slice S0(0, kDataLen);
  float* device_out;
  hipMalloc(&device_out, kDataLen * sizeof(float));
  glang::DynArray<float> Out(kDataLen, device_out);

  type_test<<<1, kDataLen + 1>>>(Out, S0);
  hipDeviceSynchronize();

  pool_test<128> <<<1, 128>>>();
  hipDeviceSynchronize();

  int hR[] = {0, 3, 4, 8, 10, 20, 26};
  float hM[256];
  auto dR = makeDA<int>(7, hR, 0);
  auto dM = makeDA<float>(256, hM, 0);
  spmv_test<6> <<<1, 6>>>(dR, dM);
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if(error != hipSuccess) {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  printf("test end.\n");
  return 0;
}
