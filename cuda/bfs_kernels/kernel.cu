#include "hip/hip_runtime.h"
/*********************************************************************************
Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

Copyright (c) 2008 International Institute of Information Technology -
Hyderabad.
All rights reserved.

Permission to use, copy, modify and distribute this software and its
documentation for
educational purpose is hereby granted without fee, provided that the above
copyright
notice and this permission notice appear in all copies of this software and that
you do
not sell the software.

THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS,
IMPLIED OR
OTHERWISE.

The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
**********************************************************************************/
#ifndef _KERNEL_H_
#define _KERNEL_H_

__device__ void clock_block(clock_t clock_count)
{
    clock_t start_clock = clock();
    clock_t clock_offset = 0;
    while (clock_offset < clock_count)
    {
        clock_offset = clock() - start_clock;
    }
}

__global__ void Kernel(Node *g_graph_nodes, int *g_graph_edges,
                       bool *g_graph_mask, bool *g_updating_graph_mask,
                       bool *g_graph_visited, int *g_cost, int no_of_nodes) {
    int tid = blockIdx.x * MAX_THREADS_PER_BLOCK + threadIdx.x;
    if (tid < no_of_nodes && g_graph_mask[tid]) {
        clock_block(100);
        g_graph_mask[tid] = false;
        for (int i = g_graph_nodes[tid].starting;
             i < (g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting);
             i++) {
            clock_block(1e4);
            int id = g_graph_edges[i];
            if (!g_graph_visited[id]) {
                g_cost[id] = g_cost[tid] + 1;
                g_updating_graph_mask[id] = true;
            }
        }
    }
}

__global__ void Kernel_ir(Node *g_graph_nodes, int *g_graph_edges,
                           bool *g_graph_mask, bool *g_updating_graph_mask,
                           bool *g_graph_visited, int *g_cost, int no_of_nodes) {
    __shared__ int task_q[MAX_THREADS_PER_BLOCK];
    __shared__ int head;

    if (threadIdx.x == 0) {
      head = 0;
    }
    __syncthreads();

    int tid = blockIdx.x * MAX_THREADS_PER_BLOCK + threadIdx.x;
    bool cond = (tid < no_of_nodes && g_graph_mask[tid]);
    int loc = 0;
    if (cond) {
      loc = atomicAdd(&head, 1);
      task_q[loc] = tid;
    }
    __syncthreads();

    if (threadIdx.x < head) {
        tid = task_q[threadIdx.x];
        g_graph_mask[tid] = false;
        for (int i = g_graph_nodes[tid].starting;
             i < (g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting);
             i++) {
            clock_block(1e4);
            int id = g_graph_edges[i];
            if (!g_graph_visited[id]) {
                g_cost[id] = g_cost[tid] + 1;
                g_updating_graph_mask[id] = true;
            }
        }
    }
}

#endif
